#include "hip/hip_runtime.h"
#include "kmeansCuda.h"
__device__
double distanceCuda(double x0,double y0, double x1, double y1){
    return sqrt(pow(x1 - x0,2) + pow(y1 - y0,2));
}

__global__
void computeClusterPoints(double* centroidx, double* centroidy, int ci, double* xs, double* ys, double* minDists, int* clusters){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    double d = distanceCuda(*centroidx,*centroidy,xs[i],ys[i]);
    //The point is near the centroid
    if(minDists[i] > d){
        //Set assigned cluster and new minimal distance
        clusters[i] = ci;
        minDists[i] = d;
    }
}

__global__
void initAcc(double* accX, double* accY){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    accX[i] = 0;
    accY[i] = 0;
}
__global__
void computeCentroids(double* accX, double* accY, int* clusters, double* xs, double* ys, double* cx, double* cy, int n, int nci){
    for(int pi = 0;pi<n;pi++){
        accX[clusters[pi]] += xs[pi];
        accY[clusters[pi]] += ys[pi];
    }

    for (int ci = 0;ci< nci;ci++){
        //Set new centroid position
        cx[ci] = accX[ci]/n;
        cy[ci] = accY[ci]/n;
    }
}


void kmeans_SOA_cuda(double** dataset,int n,int k, double** centroids,int nci,int epochs, double* bounds){
    SOAPoint* points ;
    
    hipMalloc((void**) &points, n*sizeof(SOAPoint));
    double* xs,*ys,*minDists, *xsHost, *ysHost, *minDistsHost;
    int* clusters, *clustersHost;

    xsHost = (double *) malloc(n*sizeof(double));
    ysHost = (double *) malloc(n*sizeof(double));
    minDistsHost = (double *) malloc(n*sizeof(double));
    clustersHost = (int *) malloc(n*sizeof(int));
    //points->xs = dataset[0];
    //points->ys = dataset[1];
    hipMalloc((void**) &(xs),n*sizeof(double));
    hipMalloc((void**) &(ys),n*sizeof(double));
    hipMemcpy(xs, dataset[0],(n)*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(ys, dataset[1],(n)*sizeof(double),hipMemcpyHostToDevice);
    hipMalloc((void**) &(minDists),n*sizeof(double));
    hipMalloc((void**) &(clusters),n*sizeof(int));

    //hipMalloc((void**)&cudaImage, imageSize);
    double* cx;
    double* cy;
    hipMalloc((void**) &cx,(n/2)*sizeof(double));
    hipMalloc((void**) &cy,(n/2)*sizeof(double));
    hipMemcpy(cx, centroids[0],(n/2)*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(cy, centroids[1],(n/2)*sizeof(double),hipMemcpyHostToDevice);



    for(int e = 0;e<epochs;e++){
        //Define accumulators to calculate x,y means of the points assigned to the k centroids
        for (int ci = 0;ci< nci;ci++){
            //Distanc of current point to current evaluated cluster
            computeClusterPoints<<<n/256,256>>>(cx,cy,ci,xs,ys,minDists,clusters);
        }

        hipMemcpy(cx, centroids[0],(n/2)*sizeof(double),hipMemcpyHostToDevice);
        hipMemcpy(cy, centroids[1],(n/2)*sizeof(double),hipMemcpyHostToDevice);

       double *accX, *accY;
       hipMalloc((void**) &accX,(n)*sizeof(double));
       hipMalloc((void**) &accY,(n)*sizeof(double));
       initAcc<<<n/128,128>>>(accX,accY);
       computeCentroids<<<1,1>>>(accX, accY,clusters, xs, ys,  cx,  cy, n, nci);

    }
}